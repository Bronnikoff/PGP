#include "hip/hip_runtime.h"
// Made by Max Bronnikov
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/iterator/permutation_iterator.h>
#include <iostream>
#include <map>
#include <string>

using namespace std;
using namespace thrust;

const unsigned BLOCKS = 256;
const unsigned THREADS = 256;


void throw_on_cuda_error(const hipError_t& code, int itter){
    if(code != hipSuccess){
        string err = hipGetErrorString(code);
        err += ", on iteration: ";
        err += to_string(itter);
        throw runtime_error(err);
    }
}

__global__ void gauss_step(double* C, unsigned* p, unsigned n, unsigned col, double max_elem){
    unsigned thrd_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned thrd_step = blockDim.x * gridDim.x;

    unsigned limit = n*n;
    for(unsigned index = thrd_idx; index < limit; index += thrd_step){
        unsigned j = index / n; // column in C
        unsigned virt_i = index - n*j;
        unsigned real_i = p[virt_i]; // row in C defined by permutation matrix
    
        if(j < col || virt_i <= col){
            continue;
        }

        double koeff = C[n*col + real_i] / max_elem;
        // if j == col => update L
        if(j == col){
            C[n*j + real_i] = koeff;
        }else{ // else update U
            C[n*j + real_i] -= koeff * C[n*j + p[col]];
        }
    }
}




int main(){
    unsigned n;
    cin >> n;
    // alloc mem to union matrix(see wiki algorithm)
    host_vector<double> h_C(n * n);
    device_vector<double> d_C(n * n);
    host_vector<unsigned> h_ansvec(n);
    host_vector<unsigned> h_p(n);
    device_vector<unsigned> d_p(n);

    // input of matrix
    for(unsigned i = 0; i < n; ++i){
        h_ansvec[i] = i; // init of permutation vector
        for(unsigned j = 0; j < n; ++j){
            cin >> h_C[j*n + i]; // we store need matrix in  transpose format here for easy thrust search
        }
    }

    // transporting mem to device:
    d_p = h_ansvec;
    d_C = h_C;

    // pointers to mem:
    double* raw_C = thrust::raw_pointer_cast(d_C.data());
    unsigned* raw_p = thrust::raw_pointer_cast(d_p.data());

    // compute  LU
    try{
        for(unsigned i = 0; i < n - 1; ++i){
            // search index of max elem in col
            auto it_beg = make_permutation_iterator(d_C.begin() + i*n, d_p.begin());
            auto it_end = make_permutation_iterator(d_C.begin() + i*n, d_p.end());

            auto max_elem = max_element(it_beg + i, it_end);
            unsigned max_idx = max_elem - it_beg;
            double max_val = *max_elem;

            //swap(d_p[i], d_p[max_idx])
            {
                unsigned temp = d_p[i];
                d_p[i] = d_p[max_idx];
                d_p[max_idx] = temp;
            }

            h_ansvec[i] = max_idx;

            gauss_step<<<BLOCKS, THREADS>>>(raw_C, raw_p, n, i, max_val);
            throw_on_cuda_error(hipGetLastError(), i);

            throw_on_cuda_error(hipDeviceSynchronize(), i);
        }
    }catch(runtime_error& err){
        cout << "ERROR: " << err.what() << endl;
    }

    h_C = d_C;
    h_p = d_p;

    /*
    // get true order for output:
    map<unsigned, unsigned> order;
    for(unsigned i = 0; i < n; ++i){
        order[h_p[i]] = i;
    }
    */

    // output for matrix:
    cout << std::scientific << std::setprecision(10);
    for(unsigned i = 0; i < n; ++i){
        for(unsigned j = 0; j < n; ++j){
            if(j){
                cout << " ";
            }

            cout << h_C[j*n + h_p[i]];
        }
        cout << endl;
    }
    // output of vector
    for(unsigned i = 0; i < n; ++i){
        if(i){
            cout << " ";
        }
        cout << h_ansvec[i];
    }
    cout << endl;

    return 0;
}