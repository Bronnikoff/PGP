
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(int i) {
	int jdx = blockIdx.x;			// Абсолютный номер потока
    int idx = threadIdx.x;					// Общее кол-во потоков
    if(!i){
        for(int j = 0; j < 100000000000; ++j){
            continue;
        }
    }
    printf("[%d, %d] = %d", idx, jdx, i);
}

int main() {

	for(int i = 0; i < 3; i++){
        printf("start itter: %d \n", i);
        kernel<<<2, 2>>>(i);
    }
        
	printf("\n");
	return 0;
}