#include "hip/hip_runtime.h"
// Made by Max Bronnikov
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/iterator/permutation_iterator.h>
#include <iostream>
#include <map>
#include <string>
#include <iomanip>

using namespace std;
using namespace thrust;

const unsigned BLOCKS = 256;
const unsigned THREADS = 256;


void throw_on_cuda_error(const hipError_t& code, int itter){
    if(code != hipSuccess){
        string err = hipGetErrorString(code);
        err += ", on iteration: ";
        err += to_string(itter);
        throw runtime_error(err);
    }
}

__global__ void gauss_step_L(double* C, unsigned* p, unsigned n, unsigned col, double max_elem){
    unsigned thrd_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned thrd_step = blockDim.x * gridDim.x;

    for(unsigned index = thrd_idx + col + 1; index < n; index += thrd_step){
        unsigned real_i = p[index]; // row in C defined by permutation matrix
        C[n*col + real_i] /= max_elem;
    }
}

__global__ void gauss_step_U(double* C, unsigned* p, unsigned n, unsigned col, double max_elem){
    unsigned thrd_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned thrd_step = blockDim.x * gridDim.x;

    unsigned limit = n*n;
    for(unsigned index = thrd_idx; index < limit; index += thrd_step){
        unsigned j = index / n; // column in C
        unsigned virt_i = index - n*j;
        unsigned real_i = p[virt_i]; // row in C defined by permutation matrix
    
        if(j <= col || virt_i <= col){
            continue;
        }

        C[n*j + real_i] -= C[n*col + real_i] * C[n*j + p[col]];
    }
}




int main(){
    unsigned n;
    cin >> n;
    // alloc mem to union matrix(see wiki algorithm)
    host_vector<double> h_C(n * n);
    device_vector<double> d_C(n * n);
    host_vector<unsigned> h_ansvec(n);
    host_vector<unsigned> h_p(n);
    device_vector<unsigned> d_p(n);

    // input of matrix
    for(unsigned i = 0; i < n; ++i){
        h_ansvec[i] = i; // init of permutation vector
        for(unsigned j = 0; j < n; ++j){
            cin >> h_C[j*n + i]; // we store need matrix in  transpose format here for easy thrust search
        }
    }

    // transporting mem to device:
    d_p = h_ansvec;
    d_C = h_C;

    // pointers to mem:
    double* raw_C = thrust::raw_pointer_cast(d_C.data());
    unsigned* raw_p = thrust::raw_pointer_cast(d_p.data());

    // compute  LU
    try{
        for(unsigned i = 0; i < n - 1; ++i){
            // search index of max elem in col
            auto it_beg = make_permutation_iterator(d_C.begin() + i*n, d_p.begin());
            auto it_end = make_permutation_iterator(d_C.begin() + i*n, d_p.end());

            auto max_elem = thrust::max_element(it_beg + i, it_end);
            unsigned max_idx = max_elem - it_beg;
            double max_val = *max_elem;

            //swap(d_p[i], d_p[max_idx])
            {
                unsigned temp = d_p[i];
                d_p[i] = d_p[max_idx];
                d_p[max_idx] = temp;
            }

            gauss_step_L<<<BLOCKS, THREADS>>>(raw_C, raw_p, n, i, max_val);
            throw_on_cuda_error(hipGetLastError(), i);

            gauss_step_U<<<BLOCKS, THREADS>>>(raw_C, raw_p, n, i, max_val);

            h_ansvec[i] = max_idx;
            
            throw_on_cuda_error(hipGetLastError(), i);

            //throw_on_cuda_error(hipDeviceSynchronize(), i);
        }
    }catch(runtime_error& err){
        cout << "ERROR: " << err.what() << endl;
    }

    h_C = d_C;
    h_p = d_p;

    // output for matrix:
    cout << std::scientific << std::setprecision(10);
    for(unsigned i = 0; i < n; ++i){
        for(unsigned j = 0; j < n; ++j){
            if(j){
                cout << " ";
            }

            cout << h_C[j*n + h_p[i]];
        }
        cout << endl;
    }
    // output of vector
    for(unsigned i = 0; i < n; ++i){
        if(i){
            cout << " ";
        }
        cout << h_ansvec[i];
    }
    cout << endl;

    return 0;
}