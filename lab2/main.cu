#include <iostream>
#include <iomanip>
#include <fstream>
#include <string>
#include "cuda_image.h"

using namespace std;

int main(){
    string path1, path2;
    CUDAImage img;
    cin >> path1 >> path2;
    ifstream fin(path1);
    ofstream fout(path2);
    cin >> img;
    img.FilterImg();
    cout << img;

    fout.close();
    fin.close();
    return 0;
}
